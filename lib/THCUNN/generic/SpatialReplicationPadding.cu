
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialReplicationPadding.cu"
#else

void THNN_(SpatialReplicationPadding_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int padL, int padR,
           int padT, int padB) {
  THArgCheck(TensorUtils<THCTensor>::canUse32BitIndexMath(state, input), 2,
             "input tensor must fit into 32-bit index math");

  int planeDim = 0;
  int dimh = 1;
  int dimw = 2;
  int numBatch = 1;

  int numInputDims = THCTensor_(nDimension)(state, input);
  THArgCheck(numInputDims == 3 || numInputDims == 4, 2,
             "input must be 3 or 4-dimensional");

  if (numInputDims == 4) {
    numBatch = THCTensor_(size)(state, input, 0);
    planeDim++;
    dimh++;
    dimw++;
  }

  int numPlanes = THCTensor_(size)(state, input, planeDim);
  int inputH = THCTensor_(size)(state, input, dimh);
  int inputW = THCTensor_(size)(state, input, dimw);
  int outputH = inputH + padT + padB;
  int outputW  = inputW + padL + padR;

  THCDeviceTensor<real, 4> devInput;
  THCDeviceTensor<real, 4> devOutput;

  if (numInputDims == 3) {
    THCTensor_(resize3d)(state, output, numPlanes, outputH, outputW);

    devInput = toDeviceTensor<real, 3>(state, input).upcastOuter<4>();
    devOutput = toDeviceTensor<real, 3>(state, output).upcastOuter<4>();
  } else {
    THCTensor_(resize4d)(state, output, numBatch, numPlanes, outputH, outputW);

    devInput = toDeviceTensor<real, 4>(state, input);
    devOutput = toDeviceTensor<real, 4>(state, output);
  }

  int outputPlaneSize = devOutput.getSize(2) * devOutput.getSize(3);
  dim3 gridSize(THCCeilDiv(outputPlaneSize, 256),
            devOutput.getSize(1),
            devOutput.getSize(0));
  dim3 blockSize(outputPlaneSize > 256 ? 256 : outputPlaneSize);

  SpatialReplicationPadding_updateOutput<<<gridSize, blockSize, 0, THCState_getCurrentStream(state)>>>(
    devInput, devOutput, padT, padB, padL, padR);

}

void THNN_(SpatialReplicationPadding_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int padL, int padR,
           int padT, int padB) {

  THArgCheck(TensorUtils<THCTensor>::canUse32BitIndexMath(state, input), 2,
                "input tensor must fit into 32-bit index math");
  THArgCheck(TensorUtils<THCTensor>::canUse32BitIndexMath(state, gradOutput), 3,
                "output gradient tensor must fit into 32-bit index math");

  int planeDim = 0;
  int dimh = 1;
  int dimw = 2;

  int numInputDims = THCTensor_(nDimension)(state, input);
  if (numInputDims == 4) {
    planeDim++;
    dimh++;
    dimw++;
  }

  THCTensor_(resizeAs)(state, gradInput, input);
  THCTensor_(zero)(state, gradInput);

  THCDeviceTensor<real, 4> devGradInput;
  THCDeviceTensor<real, 4> devGradOutput;

  if (numInputDims == 3) {
    devGradInput = toDeviceTensor<real, 3>(state, gradInput).upcastOuter<4>();
    devGradOutput = toDeviceTensor<real, 3>(state, gradOutput).upcastOuter<4>();
  } else {
    devGradInput = toDeviceTensor<real, 4>(state, gradInput);
    devGradOutput = toDeviceTensor<real, 4>(state, gradOutput);
  }

  int outputPlaneSize = devGradOutput.getSize(2) * devGradOutput.getSize(3);
  dim3 gridSize(THCCeilDiv(outputPlaneSize, 256),
            devGradOutput.getSize(1),
            devGradOutput.getSize(0));
  dim3 blockSize(outputPlaneSize > 256 ? 256 : outputPlaneSize);

  SpatialReplicationPadding_updateGradInput<<<gridSize, blockSize, 0, THCState_getCurrentStream(state)>>>(
    devGradInput, devGradOutput, padT, padB, padL, padR);

}

#endif