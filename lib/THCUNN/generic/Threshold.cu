
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/Threshold.cu"
#else

#include "../common.h"

void THNN_(Threshold_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           real threshold,
           real val,
           bool inplace)
{
  THCUNN_assertSameGPU_generic(state, 2, input, output);

  if (inplace)
  {
    THC_pointwiseApply1(state, input,
      ThresholdUpdateOutputIP<real>(threshold, val)
    );
    THCTensor_(set)(state, output, input);
  }
  else
  {
    THCTensor_(resizeAs)(state, output, input);
    THC_pointwiseApply2(state, output, input,
      ThresholdUpdateOutput<real>(threshold, val)
    );
  }

  THCudaCheck(cudaGetLastError());
}

void THNN_(Threshold_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           real threshold,
           real val,
           bool inplace)
{
  THCUNN_assertSameGPU_generic(state, 3, input, gradInput, gradOutput);

  if (inplace)
  {
    THC_pointwiseApply2(state, gradOutput, input,
      ThresholdUpdateGradInputIP<real>(threshold)
    );
    THCTensor_(set)(state, gradInput, gradOutput);
  }
  else
  {
    THCTensor_(resizeAs)(state, gradInput, input);
    THC_pointwiseApply3(state, gradInput, input, gradOutput,
       ThresholdUpdateGradInput<real>(threshold)
    );
  }

  THCudaCheck(cudaGetLastError());
}

#endif
