
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/MarginCriterion.cu"
#else

void THNN_(MarginCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           bool sizeAverage,
           real margin)
{
  THCUNN_assertSameGPU_generic(state, 2, input, target);

  long size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));
  accreal sum = thrust::inner_product(input_data, input_data+size, target_data, (accreal) 0, thrust::plus<accreal>(),
      margin_functor<real, accreal>(ScalarConvert<real, accreal>::to(margin)));

  if (sizeAverage)
    sum /= size;

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);

  THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, real>::to(sum));
}


void THNN_(MarginCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradInput,
           bool sizeAverage,
           real margin)
{
  THCUNN_assertSameGPU_generic(state, 3, input, target, gradInput);

  long size = THCTensor_(nElement)(state, input);
  accreal norm = sizeAverage ? 1.f/size : 1;

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  THCTensor_(resizeAs)(state, gradInput, input);

  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));
  thrust::device_ptr<real> gradInput_data(THCTensor_(data)(state, gradInput));

  thrust::transform(input_data, input_data+size, target_data, gradInput_data,
      margin_updateGradInput_functor<real, accreal>(ScalarConvert<real, accreal>::to(margin), norm));

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
}

#endif