
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/Tanh.cu"
#else

#include "../common.h"

void THNN_(Tanh_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output)
{
  THCUNN_assertSameGPU_generic(state, 2, input, output);
  THCTensor_(resizeAs)(state, output, input);
  THC_pointwiseApply2(state, output, input, tanhupdateOutput_functor<real>());
}

void THNN_(Tanh_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *output)
{
  THCUNN_assertSameGPU_generic(state, 3, output, gradOutput, gradInput);
  THCTensor_(resizeAs)(state, gradInput, output);
  THC_pointwiseApply3(state, gradInput, output, gradOutput, tanhupdateGradInput_functor<real>());
}

#endif
