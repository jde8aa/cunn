
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/VolumetricAveragePooling.cu"
#else

void THNN_(VolumetricAveragePooling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int kT, int kW, int kH,
           int dT, int dW, int dH)
{
  int batchSize;
  int inputSlices;
  int inputTime;
  int inputHeight;
  int inputWidth;

  if (THCTensor_(nDimension)(state, input) == 4)
  {
    THArgCheck(
      THCTensor_(size)(state, input, 1) >= kT &&
      THCTensor_(size)(state, input, 2) >= kH &&
      THCTensor_(size)(state, input, 3) >= kW, 2,
      "input image smaller than kernel size"
    );

    /* sizes */
    batchSize   = 1;
    inputSlices = THCTensor_(size)(state, input, 0);
    inputTime   = THCTensor_(size)(state, input, 1);
    inputHeight = THCTensor_(size)(state, input, 2);
    inputWidth  = THCTensor_(size)(state, input, 3);
  }
  else if (THCTensor_(nDimension)(state, input) == 5)
  {
    THArgCheck(
      THCTensor_(size)(state, input, 2) >= kT &&
      THCTensor_(size)(state, input, 3) >= kH &&
      THCTensor_(size)(state, input, 4) >= kW, 2,
      "input image smaller than kernel size"

    );
    /* sizes */
    batchSize   = THCTensor_(size)(state, input, 0);
    inputSlices = THCTensor_(size)(state, input, 1);
    inputTime   = THCTensor_(size)(state, input, 2);
    inputHeight = THCTensor_(size)(state, input, 3);
    inputWidth  = THCTensor_(size)(state, input, 4);
  }
  else
  {
    THArgCheck(false, 2, "4D or 5D tensor expected");
  }

  int outputTime   = (inputTime   - kT) / dT + 1;
  int outputHeight = (inputHeight - kH) / dH + 1;
  int outputWidth  = (inputWidth  - kW) / dW + 1;

  if (input->nDimension == 4) /* 4D */
  {
    /* resize output */
    THCTensor_(resize4d)(state, output, inputSlices,
                         outputTime, outputHeight, outputWidth);
  }
  else /* 5D */
  {
    THCTensor_(resize5d)(state, output, batchSize, inputSlices,
                         outputTime, outputHeight, outputWidth);
  }

  input = THCTensor_(newContiguous)(state, input);

  // Collapse batch and feature dimensions
  THCDeviceTensor<real, 4> cudaInput;
  THCDeviceTensor<real, 4> cudaOutput;
  if (THCTensor_(nDimension)(state, input) == 4)
  {
    cudaInput  = toDeviceTensor<real, 4>(state, input);
    cudaOutput = toDeviceTensor<real, 4>(state, output);
  }
  else
  {
    cudaInput  = toDeviceTensor<real, 5>(state, input).downcastOuter<4>();
    cudaOutput = toDeviceTensor<real, 5>(state, output).downcastOuter<4>();
  }

  int totalZ = outputTime * inputSlices * batchSize;
  int offsetZ = 0;
  dim3 block(32, 8);
  while (totalZ > 0) {
    dim3 grid(THCCeilDiv(outputWidth, static_cast<int>(block.x)),
              THCCeilDiv(outputHeight, static_cast<int>(block.y)),
              totalZ > 65535 ? 65535 : totalZ);

    accreal normFactor = ScalarConvert<int, accreal>::to(1) / static_cast<accreal>(kT * kH * kW);
    switch (kW)
      {
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(1);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(2);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(3);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(4);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(5);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(6);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(7);
      default:
        cuda_VolumetricAveragePooling_updateOutput<real, accreal><<<grid, block>>>(
                                                                    cudaInput,
                                                                    cudaOutput,
                                                                    kT, kH, kW,
                                                                    dT, dH, dW,
                                                                    normFactor,
                                                                    offsetZ
                                                                    );
        break;
      }
    totalZ -= 65535;
    offsetZ += 65535;
    THCudaCheck(cudaGetLastError());
  }
  THCTensor_(free)(state, input);
}

void THNN_(VolumetricAveragePooling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int kT, int kW, int kH,
           int dT, int dW, int dH)
{
  bool kernelsOverlap = (dT < kT) || (dH < kH) || (dW < kW);

  // Resize and initialize result tensor.
  THCTensor_(resizeAs)(state, gradInput, input);
  THCTensor_(zero)(state, gradInput);

  int batchSize;
  int inputSlices;
  int inputTime;
  int inputHeight;
  int inputWidth;

  int outputTime;
  int outputHeight;
  int outputWidth;

  if (THCTensor_(nDimension)(state, input) == 4) /* 4D */
  {
    batchSize = 1;
    inputSlices  = THCTensor_(size)(state, input, 0);
    inputTime    = THCTensor_(size)(state, input, 1);
    inputHeight  = THCTensor_(size)(state, input, 2);
    inputWidth   = THCTensor_(size)(state, input, 3);

    outputTime   = THCTensor_(size)(state, gradOutput, 1);
    outputHeight = THCTensor_(size)(state, gradOutput, 2);
    outputWidth  = THCTensor_(size)(state, gradOutput, 3);
  }
  else
  {
    batchSize    = THCTensor_(size)(state, input, 0);
    inputSlices  = THCTensor_(size)(state, input, 1);
    inputTime    = THCTensor_(size)(state, input, 2);
    inputHeight  = THCTensor_(size)(state, input, 3);
    inputWidth   = THCTensor_(size)(state, input, 4);

    outputTime   = THCTensor_(size)(state, gradOutput, 2);
    outputHeight = THCTensor_(size)(state, gradOutput, 3);
    outputWidth  = THCTensor_(size)(state, gradOutput, 4);
  }

  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  // Collapse batch and feature dimensions
  THCDeviceTensor<real, 4> cudaGradInput;
  THCDeviceTensor<real, 4> cudaGradOutput;
  if (THCTensor_(nDimension)(state, input) == 4)
  {
    cudaGradInput  = toDeviceTensor<real, 4>(state, gradInput);
    cudaGradOutput = toDeviceTensor<real, 4>(state, gradOutput);
  }
  else
  {
    cudaGradInput =
      toDeviceTensor<real, 5>(state, gradInput).downcastOuter<4>();
    cudaGradOutput =
      toDeviceTensor<real, 5>(state, gradOutput).downcastOuter<4>();
  }

  dim3 block(32, 8);

  // Optimizing for stride 1 is probably only of limited value, but this
  // specialization yields 3x speedup over the atomicAdd implementation.
  if (dT == 1 && dH == 1 && dW == 1)
  {
    int totalZ = inputTime * inputSlices * batchSize;
    int offsetZ = 0;
    while (totalZ > 0) {
      dim3 grid(THCCeilDiv(inputWidth, static_cast<int>(block.x)),
                THCCeilDiv(inputHeight, static_cast<int>(block.y)),
                totalZ > 65535 ? 65535 : totalZ);
      cuda_VolumetricAveragePooling_updateGradInput_Stride1<real, accreal><<<grid, block>>>(
         cudaGradOutput, cudaGradInput, kT, kH, kW, 1.0f/(kT * kH * kW), offsetZ);
      THCudaCheck(cudaGetLastError());
      totalZ -= 65535;
      offsetZ += 65535;
    }
  }
  else
  {
    int totalZ = outputTime * inputSlices * batchSize;
    int offsetZ = 0;
    while (totalZ > 0) {

      dim3 grid(THCCeilDiv(outputWidth, static_cast<int>(block.x)),
                THCCeilDiv(outputHeight, static_cast<int>(block.y)),
                totalZ > 65535 ? 65535 : totalZ);
      if (kernelsOverlap)
        {
          cuda_VolumetricAveragePooling_updateGradInput_atomicAdd<real, accreal><<<grid, block>>>(
            cudaGradOutput, cudaGradInput, kT, kH, kW, dT, dH, dW, offsetZ);
        }
      else
        {
          cuda_VolumetricAveragePooling_updateGradInput<real, accreal><<<grid, block>>>(
             cudaGradOutput, cudaGradInput, kT, kH, kW, dT, dH, dW, offsetZ);
        }
      THCudaCheck(cudaGetLastError());
      totalZ -= 65535;
      offsetZ += 65535;
    }
  }

  THCTensor_(free)(state, gradOutput);
}

#endif
